#include "hip/hip_runtime.h"

//          Copyright RedPortal 2017 - 2017.
// Distributed under the Boost Software License, Version 1.0.
//    (See accompanying file LICENSE or copy at
//          http://www.boost.org/LICENSE_1_0.txt)

#include <mgcpp/kernels/bits/fill.cuh>
#include <cmath>

#define BLK 64

namespace mgcpp
{
    __global__  void
    mgblas_Sfill_impl(float* arr, float value, size_t n)
    {
	int const id = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float shared[64];

	if(id >= n)
	    return;

	shared[threadIdx.x] = value;
	__syncthreads();

	arr[id] = shared[threadIdx.x];
    }

    __global__  void
    mgblas_Dfill_impl(double* arr, double value, size_t n)
    {
	int const id = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ double shared[64];

	if(id >= n)
	    return;

	shared[threadIdx.x] = value;
	__syncthreads();

	arr[id] = shared[threadIdx.x];
    }

    __global__  void
    mgblas_Cfill_impl(hipComplex* arr, hipComplex value, size_t n)
    {
        int const id = blockIdx.x * blockDim.x + threadIdx.x;
        __shared__ hipComplex shared[64];

        if(id >= n)
        return;

        shared[threadIdx.x] = value;
        __syncthreads();

        arr[id] = shared[threadIdx.x];
    }

    __global__  void
    mgblas_Zfill_impl(hipDoubleComplex* arr, hipDoubleComplex value, size_t n)
    {
        int const id = blockIdx.x * blockDim.x + threadIdx.x;
        __shared__ hipDoubleComplex shared[64];

        if(id >= n)
        return;

        shared[threadIdx.x] = value;
        __syncthreads();

        arr[id] = shared[threadIdx.x];
    }

    __global__  void
    mgblas_Hfill_impl(__half* arr, __half value, size_t n)
    {
        int const id = blockIdx.x * blockDim.x + threadIdx.x;
        __shared__ __half shared[64];

        if(id >= n)
        return;

        shared[threadIdx.x] = value;
        __syncthreads();

        arr[id] = shared[threadIdx.x];
    }

    kernel_status_t
    mgblas_Sfill(float* arr, float value, size_t n)
    {
	int grid_size = static_cast<int>(
	    ceil(static_cast<float>(n)/ BLK ));
	mgblas_Sfill_impl<<<BLK, grid_size>>>(arr, value, n);

	return success;
    }

    kernel_status_t
    mgblas_Dfill(double* arr, double value, size_t n)
    {
	int grid_size = static_cast<int>(
	    ceil(static_cast<float>(n)/ BLK ));
	mgblas_Dfill_impl<<<BLK, grid_size>>>(arr, value, n);

	return success;
    }

    kernel_status_t
    mgblas_Cfill(hipComplex* arr, hipComplex value, size_t n)
    {
        int grid_size = static_cast<int>(
            ceil(static_cast<float>(n)/ BLK ));
        mgblas_Cfill_impl<<<BLK, grid_size>>>(arr, value, n);

        return success;
    }

    kernel_status_t
    mgblas_Zfill(hipDoubleComplex* arr, hipDoubleComplex value, size_t n)
    {
        int grid_size = static_cast<int>(
            ceil(static_cast<float>(n)/ BLK ));
        mgblas_Zfill_impl<<<BLK, grid_size>>>(arr, value, n);

        return success;
    }

    kernel_status_t
    mgblas_Hfill(__half* arr, __half value, size_t n)
    {
        int grid_size = static_cast<int>(
            ceil(static_cast<float>(n)/ BLK ));
        mgblas_Hfill_impl<<<BLK, grid_size>>>(arr, value, n);

        return success;
    }
}
